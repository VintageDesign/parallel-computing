#include "hip/hip_runtime.h"
#include "floyd_cuda.h"
#include "utils.h"


__global__ void shortest_path_kernel(int k, int node_count, int * matrix_cuda)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i = 0; i < node_count; i++)
    {
        if(matrix_cuda[i * node_count + index] > matrix_cuda[i * node_count + k] + matrix_cuda[k * node_count + index])
        {
            matrix_cuda[i * node_count + index] = matrix_cuda[i * node_count + k] + matrix_cuda[k * node_count + index];
        }
    }

}


void floyd_cuda(int node_count, int *** adj_matrix)
{
    int * matrix_cuda;
    
    // Copy to GPU
    hipMalloc((void **) &matrix_cuda,node_count *  node_count * sizeof(int));
    
    for(int i = 0; i < node_count; i++)
    {
        hipMemcpy(&matrix_cuda[i * node_count], (*adj_matrix)[i], (size_t) node_count * sizeof(int), hipMemcpyHostToDevice);
    }


    // do the thing
    for(int k = 0; k < node_count; k++)
    {
        shortest_path_kernel <<<node_count, 1>>>(k,node_count, matrix_cuda);
        hipDeviceSynchronize();
    }
   



    // Copy back to Host
    for(int i = 0; i < node_count; i++)
    {
        hipMemcpy((*adj_matrix)[i], &matrix_cuda[i * node_count], (size_t) node_count * sizeof(int), hipMemcpyDeviceToHost);
    }

}

